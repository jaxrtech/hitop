#include "hip/hip_runtime.h"
#include "crossover_method.cuh"

#include "util.cuh"

namespace hitop {
namespace algo {
namespace crossover_method {

typedef typename detail::ProgramDesciptorPair ProgramDesciptorPair;

__host__ __device__
ProgramDesciptorPair::ProgramDesciptorPair(const ProgramDescriptor& a_, const ProgramDescriptor& b_)
	: a(a_)
	, b(b_)
{ }

//

point_crossover::point_crossover(thrust::device_ptr<algo::Settings> settings,
								 thrust::device_ptr<ProgramDescriptor> descriptors_old,
								 thrust::device_ptr<uint8_t> pool_old,
								 thrust::device_ptr<ProgramDescriptor> descriptors_new,
								 thrust::device_ptr<uint8_t> pool_new,
								 size_t pool_length)

	: settings_(settings.get())
	, descriptors_old_(descriptors_old.get())
	, descriptors_new_(descriptors_new.get())
	, pool_old_(pool_old.get())
	, pool_new_(pool_new.get())
	, pool_length_(pool_length)
{
	rng_ = thrust::default_random_engine(util::get_entropy_with_ptr(this));
}

__host__ __device__
ProgramDesciptorPair point_crossover::get_descriptor_pair(const SelectionResult& result)
{
	if (result.mode != SelectionResult::Mode::Pair) {
		printf("error: get_descriptor_pair() called with result of `Mode::Single`");
		// TODO: how to signal error from __device__ function
	}

	size_t a_idx = result.parent_index_a;
	size_t b_idx = result.parent_index_b;

	ProgramDescriptor& a_descriptor = descriptors_old_[a_idx];
	ProgramDescriptor& b_descriptor = descriptors_old_[b_idx];

	return ProgramDesciptorPair(a_descriptor, b_descriptor);
}

__host__ __device__
size_t point_crossover::get_child_length(const SelectionResult& result)
{
	switch (result.mode)
	{
	case SelectionResult::Mode::Single: {
		auto i = result.single_index;

		ProgramDescriptor descriptor = descriptors_old_[i];
		return descriptor.length;
	}

	case SelectionResult::Mode::Pair: {
		auto pair = get_descriptor_pair(result);

		size_t min_length = thrust::min(pair.a.length, pair.b.length);
		size_t max_length = thrust::max(pair.a.length, pair.b.length);

		thrust::uniform_int_distribution<size_t> length_range(min_length, max_length);
		size_t child_length = length_range(rng_);

		return child_length;
	}

	default: {
		printf("error: invalid SelectionResult::Mode enum value");
		return 0;
	}

	}
}

__host__ __device__
void point_crossover::operator()(SelectionResult& result)
{
	switch (result.mode)
	{
	case SelectionResult::Mode::Single: {
		size_t descriptor_idx = result.single_index;
		auto& descriptor = descriptors_old_[descriptor_idx];

		size_t pool_pos = settings_->block_size * result.destination_index;

		// Copy old program to new pool since there is nothing to cross over with

		// TODO: We are assuming the pool position will be the same
		size_t count = 0;
		size_t i = pool_pos;

		while (i < pool_length_
				&& count < descriptor.length) {

			pool_new_[i] = pool_old_[i];

			//

			i++;
			count++;
		}

		descriptors_new_[result.destination_index] = descriptors_old_[descriptor_idx];

		break;
	}

	case SelectionResult::Mode::Pair: {
		auto pair = get_descriptor_pair(result);
		size_t child_length = get_child_length(result);
		size_t child_pool_pos = settings_->block_size * result.destination_index;

		thrust::uniform_real_distribution<float> r01(0.0, 1.0);

		size_t count = 0;
		size_t a_idx = pair.a.pos;
		size_t b_idx = pair.b.pos;
		size_t x_idx = child_pool_pos;

		// Will be +1 from last element in array
		size_t end_idx = pool_length_;

		while (a_idx < end_idx
				&& b_idx < end_idx
				&& x_idx < end_idx
				&& count < child_length) {

			// Select parent randomly
			uint8_t value;
			if (r01(rng_) < 0.5) {
				value = pool_old_[a_idx];
			}
			else {
				value = pool_old_[b_idx];
			}

			pool_new_[x_idx] = value;

			//

			a_idx++;
			b_idx++;
			x_idx++;
			count++;
		}

		// Create the child program descriptor
		ProgramDescriptor child_descriptor;
		child_descriptor.pos = child_pool_pos;
		child_descriptor.length = child_length;
		child_descriptor.score = NAN;

		descriptors_new_[result.destination_index] = child_descriptor;
		break;
	}

	}

}


} // namespace crossover_method
} // namespace algo
} // namespace hitop