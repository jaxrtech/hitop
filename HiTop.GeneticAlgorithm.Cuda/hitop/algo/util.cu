#include "hip/hip_runtime.h"
#include "util.cuh"

#include <common_functions.h>

namespace hitop {
namespace algo {
namespace util {

__host__ __device__
uint32_t hash32(uint32_t a)
{
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

__host__ __device__
uint64_t hash64(uint64_t key)
{
	key = (~key) + (key << 21); // key = (key << 21) - key - 1;
	key = key ^ (key >> 24);
	key = (key + (key << 3)) + (key << 8); // key * 265
	key = key ^ (key >> 14);
	key = (key + (key << 2)) + (key << 4); // key * 21
	key = key ^ (key >> 28);
	key = key + (key << 31);
	return key;
}

__host__ __device__
uint32_t hash64_32(uint64_t key)
{
	key = (~key) + (key << 18); // key = (key << 18) - key - 1;
	key = key ^ (key >> 31);
	key = key * 21; // key = (key + (key << 2)) + (key << 4);
	key = key ^ (key >> 11);
	key = key + (key << 6);
	key = key ^ (key >> 22);
	return static_cast<uint32_t>(key);
}

__host__ __device__
uint32_t get_entropy_with_ptr(const void* address)
{
	auto entropy = reinterpret_cast<uintptr_t>(address);
	uint32_t seed = util::hash32(entropy) ^ util::hash32(clock());

	return seed;
}

} // namespace util
} // namespace algo
} // namespace hitop