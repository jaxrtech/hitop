#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>

#include <thrust/device_new.h>
#include <thrust/device_delete.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/sort.h>
#include <thrust/logical.h>
#include <thrust/random.h>

#include "util.h"

// Monkey patch onto thrust
namespace thrust {

template<typename T>
thrust::device_ptr<T> device_new(const T& host_ptr)
{
	return thrust::device_new<T>(thrust::device_malloc<T>(1),
								 host_ptr);
}

}

namespace hitop {
namespace algo {

struct Settings
{
	size_t min_program_size;
	size_t max_program_size;
	size_t block_size;
};

namespace util {

__host__ __device__
uint32_t hash32(uint32_t a)
{
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

__host__ __device__
uint64_t hash64(uint64_t key)
{
	key = (~key) + (key << 21); // key = (key << 21) - key - 1;
	key = key ^ (key >> 24);
	key = (key + (key << 3)) + (key << 8); // key * 265
	key = key ^ (key >> 14);
	key = (key + (key << 2)) + (key << 4); // key * 21
	key = key ^ (key >> 28);
	key = key + (key << 31);
	return key;
}

__host__ __device__
uint32_t hash64_32(uint64_t key)
{
	key = (~key) + (key << 18); // key = (key << 18) - key - 1;
	key = key ^ (key >> 31);
	key = key * 21; // key = (key + (key << 2)) + (key << 4);
	key = key ^ (key >> 11);
	key = key + (key << 6);
	key = key ^ (key >> 22);
	return static_cast<uint32_t>(key);
}

__host__ __device__
uint32_t get_entropy_with_ptr(const void* address)
{
	auto entropy = reinterpret_cast<uintptr_t>(address);
	uint32_t seed = util::hash32(entropy) ^ util::hash32(clock());

	return seed;
}

} // namespace util

namespace program_descriptor {

typedef float score_t;

struct ProgramDescriptor
{
	size_t pos;
	size_t length;
	score_t score;
};


struct create_random : public thrust::unary_function<size_t, ProgramDescriptor>
{
private:
	Settings* settings_;

	//algo::util::RandomContext* random_context_;

public:
	create_random(thrust::device_ptr<algo::Settings> settings_ptr)
		: settings_(settings_ptr.get())
	{ }

	__host__ __device__
		ProgramDescriptor operator()(size_t index)
	{
		uint32_t seed = util::get_entropy_with_ptr(this);
		thrust::default_random_engine rng(seed);
		thrust::uniform_int_distribution<size_t> program_length(settings_->min_program_size, settings_->max_program_size);

		ProgramDescriptor descriptor;
		descriptor.pos = settings_->block_size * index;
		descriptor.length = program_length(rng);
		descriptor.score = NAN;

		return descriptor;
	}
};

struct greater_score : public thrust::binary_function<ProgramDescriptor, ProgramDescriptor, bool>
{
	__host__ __device__
		bool operator()(const ProgramDescriptor& a, const ProgramDescriptor& b)
	{
		return a.score > b.score;
	}
};

struct score : public thrust::unary_function<ProgramDescriptor, score_t>
{
	__host__ __device__
		score_t operator()(const ProgramDescriptor& x)
	{
		return x.score;
	}
};

} // namespace program_descriptor

// Import type into `algo` namespace
typedef typename program_descriptor::score_t score_t;

namespace selection_result {

enum class Mode { Single, Pair };

struct SelectionResult
{
	Mode mode;

	union {
		struct { size_t single_index; };

		struct {
			size_t parent_index_a;
			size_t parent_index_b;
		};
	};
};

__host__ __device__
SelectionResult create_single_index(const size_t index)
{
	SelectionResult result;
	result.mode = Mode::Single;
	result.single_index = index;

	return result;
}

__host__ __device__
SelectionResult create_pair_indices(const size_t parent_index_a, const size_t parent_index_b)
{
	SelectionResult result;
	result.mode = Mode::Pair;
	result.parent_index_a = parent_index_a;
	result.parent_index_b = parent_index_b;

	return result;
}

struct from_index : public thrust::unary_function<size_t, SelectionResult>
{
	__host__ __device__
		SelectionResult operator()(size_t index)
	{
		return create_single_index(index);
	}
};

} // namespace selection_result

namespace selection_method {

typedef selection_result::SelectionResult SelectionResult;
typedef program_descriptor::ProgramDescriptor ProgramDescriptor;

struct tournament_selection : public thrust::unary_function<size_t, SelectionResult>
{
	struct Context
	{
		thrust::device_ptr<ProgramDescriptor> program_descriptors;
		size_t program_descriptors_count;

		uint32_t rounds;
	};


#define DEBUG_PRINT_LINE() printf("%d\n", __LINE__);

private:
	Context *context_;

	thrust::default_random_engine rng_;
	
	typedef thrust::uniform_int_distribution<size_t> indices_range_distribution;
	indices_range_distribution indices_range_;

	__host__ __device__
	size_t get_random_index()
	{
		return indices_range_(rng_);
	}

	__host__ __device__
	size_t get_next_winner_index()
	{
		auto program_descriptors = context_->program_descriptors.get();

		auto cur_idx = get_random_index();
		auto cur_score = program_descriptors[cur_idx].score;

		for (uint32_t i = 0; i < context_->rounds; i++) {
			auto other_idx = get_random_index();
			auto other_score = program_descriptors[other_idx].score;

			if (other_score > cur_score) {
				cur_idx = other_idx;
			}
		}

		return cur_idx;
	}


public:
	tournament_selection(thrust::device_ptr<Context> context_ptr)
		: context_(context_ptr.get())
	{
		rng_ = thrust::default_random_engine(util::get_entropy_with_ptr(this));
	}

	__host__ __device__
	SelectionResult operator()(size_t index)
	{
		// Only now can we dereference `context_` since we are now dereferencing a device pointer on
		// the device versus in the constructor which is solely on the host.
		indices_range_ = indices_range_distribution(0, context_->program_descriptors_count);

		return selection_result::create_pair_indices(index, get_next_winner_index());
	}
};

} // namespace selection_method

namespace program {

struct fill : public thrust::unary_function<program_descriptor::ProgramDescriptor&, void>
{
private:
	thrust::device_ptr<uint8_t> pool_;

	thrust::default_random_engine rng_;

public:
	fill(thrust::device_ptr<uint8_t> pool)
		: pool_(pool)
	{
		rng_ = thrust::default_random_engine(util::get_entropy_with_ptr(this));
	}

	__host__ __device__
		void operator()(program_descriptor::ProgramDescriptor& descriptor)
	{
		thrust::uniform_int_distribution<uint8_t> byte_range(0, UINT8_MAX);

		for (size_t i = descriptor.pos;
			 i < descriptor.pos + descriptor.length;
			 i++) {

			pool_[i] = byte_range(rng_);
		}
	}
};

struct score : public thrust::unary_function<program_descriptor::ProgramDescriptor&, void>
{
private:
	thrust::device_ptr<uint8_t> pool;

public:
	score(thrust::device_ptr<uint8_t> pool)
		: pool(pool)
	{
	}

	__host__ __device__
		void operator()(program_descriptor::ProgramDescriptor& descriptor)
	{
		// TODO: Dummy scoring function

		uint32_t seed = hitop::algo::util::hash64_32(descriptor.pos);

		thrust::default_random_engine rng(seed);
		thrust::uniform_real_distribution<float> score_range(0, descriptor.length);

		descriptor.score = score_range(rng);
	}
};


} // namespace program

} // namespace algo
} // namespace hitop



int main(int argc, char* argv[])
{
	hitop::util::AppSettings settings;
	if (!hitop::util::try_parse_settings(argc, argv, settings)) {
		hitop::util::output_usage(std::cerr);
		return -1;
	}

	thrust::host_vector<uint8_t> target_h;
	std::streamsize target_length;
	bool result = hitop::util::try_read_file(settings.input_path, &target_h, &target_length);
	if (!result) {
		return -1;
	}

	if (target_length <= 0) {
		std::cerr << "error: todo: file is empty and will not be compressed" << std::endl;
		return -1;
	}

	// Actually do stuff
	hitop::util::output_header(std::cout);

	std::cout
		<< "info: started at " << hitop::util::get_timestamp_readable() 
		<< std::endl;

	std::cout
		<< "input file: '" << settings.input_path << std::endl
		<< "size: " << target_length
		<< std::endl
		<< std::endl;

	// Wait for CUDA to initialize
	std::cout
		<< "info: waiting for CUDA to initialize and warm up..."
		<< std::endl;

	hipFree(0);

	std::cout
		<< "info: done"
		<< std::endl
		<< std::endl;

	// Copy to device
	std::cout
		<< "debug: copying target file to device..."
		<< std::endl;

	thrust::device_vector<char> target_d = target_h;

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// Set algorithm Settings
	//

	using namespace hitop;
	using namespace hitop::algo;
	using algo::selection_result::SelectionResult;
	using algo::program_descriptor::ProgramDescriptor;

	const size_t population_count = 100;
	assert(population_count > 0);

	const size_t selection_elites = 2;

	const bool enable_stats_output = true;
	const size_t generations_per_stats_output = 1;

	const size_t program_pool_size = population_count * target_length;
	assert(program_pool_size > 0);

	size_t generation_num = 0;

	//

	algo::Settings program_settings_h;
	program_settings_h.min_program_size = target_length * (0.75);
	program_settings_h.max_program_size = target_length;
	program_settings_h.block_size = target_length;

	auto program_settings_d = thrust::device_new<algo::Settings>(program_settings_h);

	//
	// Allocate device_vectors
	//

	const size_t program_descriptors_data_size = sizeof(program_descriptor::ProgramDescriptor) * population_count;

	std::cout
		<< "debug: allocating "
		<< population_count << " program descriptors "
		<< "(" << program_descriptors_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<program_descriptor::ProgramDescriptor> program_descriptors(population_count);

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//

	const size_t program_pool_data_size = sizeof(uint8_t) * program_pool_size;

	std::cout
		<< "debug: allocating program pools "
		<< "(" << program_pool_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<uint8_t> program_pool(program_pool_size);

	thrust::device_vector<uint8_t> program_pool_temp(program_pool_size);
	
	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//

	const size_t selection_results_data_size = sizeof(SelectionResult) * program_pool_size;

	std::cout
		<< "debug: allocating selection results "
		<< "(" << selection_results_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<SelectionResult> selection_results(program_pool_size);

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// Setup selection settings now that we have allocated everything
	//

	typedef typename algo::selection_method::tournament_selection::Context SelectionContext;

	SelectionContext selection_context_h;
	selection_context_h.program_descriptors = program_descriptors.data();
	selection_context_h.program_descriptors_count = program_descriptors.size();
	selection_context_h.rounds = 5;

	auto selection_context_d = thrust::device_new<SelectionContext>(selection_context_h);

	//
	// Initialize initial generation
	//

	// Initialize program descriptors to random lengths
	std::cout
		<< "debug: initializing program descriptors to random lengths"
		<< std::endl;

	thrust::transform(thrust::counting_iterator<size_t>(0),
					  thrust::counting_iterator<size_t>(program_descriptors.size()),
					  program_descriptors.begin(),
					  program_descriptor::create_random(program_settings_d));
	
	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	// Fill program pool with random values per each program descriptor
	std::cout
		<< "debug: initializing all programs to random data"
		<< std::endl;

	thrust::for_each(program_descriptors.begin(),
					 program_descriptors.end(),
					 program::fill(program_pool.data()));

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;


	//
	// Evaluate fitness of initial generation 
	//

	thrust::for_each(program_descriptors.begin(),
					 program_descriptors.end(),
					 program::score(program_pool.data()));

	//
	// Sort program descriptors by their scores descending
	//

	using namespace thrust::placeholders;

	thrust::sort(program_descriptors.begin(),
				 program_descriptors.end(),
				 program_descriptor::greater_score());

	//
	// Calculate statistics on current scores if necessary
	//

	if (enable_stats_output
		&& generation_num % generations_per_stats_output == 0) {

		ProgramDescriptor best = *(program_descriptors.begin());
		ProgramDescriptor worst = *(program_descriptors.end() - 1);

		score_t sum = thrust::transform_reduce(program_descriptors.begin(),
					                           program_descriptors.end(),
											   program_descriptor::score(),
								               0.0f,
								               thrust::plus<program_descriptor::score_t>());

		score_t avg = sum / program_descriptors.size();

		std::cout
			<< "gen " << generation_num << ": "
			<< "best = " << best.score << " | "
			<< "avg = " << avg << " | "
			<< "worst = " << worst.score
			<< std::endl;
	}

	//
	// Selection process
	//


	//
	// Select elites if setting specified
	//

	std::cout
		<< "debug: selecting elites..."
		<< std::endl;

	// Keep track of the position to start at in the case that we use elites to skip running the
	// selection method over some of the programs
	auto selection_results_start = selection_results.begin();
	auto program_descriptors_start = program_descriptors.begin();
	auto start_index = 0;

	if (selection_elites > 0) {
		thrust::transform(thrust::counting_iterator<size_t>(0),
						  thrust::counting_iterator<size_t>(selection_elites),
						  selection_results_start,
						  selection_result::from_index());

		start_index += selection_elites;
		thrust::advance(selection_results_start, selection_elites);
		thrust::advance(program_descriptors_start, selection_elites);
	}

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// Run the selection method on the rest
	//

	std::cout
		<< "debug: selecting the rest of population with selection method..."
		<< std::endl;

	thrust::transform(thrust::counting_iterator<size_t>(start_index),
					  thrust::counting_iterator<size_t>(program_descriptors.size()),
					  selection_results_start,
					  selection_method::tournament_selection(selection_context_d));

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// Crossover selection results
	//

	// TODO

	//
	// Mutate newly generated population
	//

	// TODO

	//
	// Swap newly created population with current population vector
	//

	// TOOD

	//
	// Loop back with new generation
	//

	// TODO

	//
	// End algo
	//

	// Clear any `device_ptr`s
	
	// FIXME: Trying to delete the `thrust::device_ptr` is breaking the build
	//thrust::device_delete(program_settings_d);
	//thrust::device_delete(selection_context_d);
	//thrust::device_delete(random_context_d);

	std::cout
		<< "info: algorithm ran to completion!" << std::endl
		<< "press any key to exit..." << std::endl;

	std::cin.ignore();
	return 0;
}
