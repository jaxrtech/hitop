#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <ctime>

#include <thrust/device_new.h>
#include <thrust/device_delete.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>

#include "util.h"

bool try_read_file(std::string input_path, thrust::host_vector<uint8_t>* data, std::streamsize* size)
{
	// open the file
	std::ifstream file(input_path, std::ios::binary);
	if (!file.is_open()) {
		std::cerr << "error: cannot open input file" << std::endl;
		return false;
	}

	// prevent eating new lines in binary mode
	file.unsetf(std::ios::skipws);

	// get its size:
	file.seekg(0, std::ios::end);
	*size = file.tellg();
	file.seekg(0, std::ios::beg);

	// reserve capacity
	data->clear();
	data->reserve(*size);

	// read the data:
	data->insert(data->begin(),
			     std::istream_iterator<uint8_t>(file),
			     std::istream_iterator<uint8_t>());

	return true;
}

namespace hitop {
namespace algo {

namespace util {

__host__ __device__
uint32_t hash32(uint32_t a)
{
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

__host__ __device__
uint64_t hash64(uint64_t key)
{
	key = (~key) + (key << 21); // key = (key << 21) - key - 1;
	key = key ^ (key >> 24);
	key = (key + (key << 3)) + (key << 8); // key * 265
	key = key ^ (key >> 14);
	key = (key + (key << 2)) + (key << 4); // key * 21
	key = key ^ (key >> 28);
	key = key + (key << 31);
	return key;
}

__host__ __device__
uint32_t hash64_32(uint64_t key)
{
	key = (~key) + (key << 18); // key = (key << 18) - key - 1;
	key = key ^ (key >> 31);
	key = key * 21; // key = (key + (key << 2)) + (key << 4);
	key = key ^ (key >> 11);
	key = key + (key << 6);
	key = key ^ (key >> 22);
	return static_cast<uint32_t>(key);
}

} // namespace util

namespace program_descriptor {

struct ProgramDescriptor
{
	size_t pos;
	size_t length;
};


struct Settings
{
	size_t min_program_size;
	size_t max_program_size;
	size_t block_size;
};

struct create_random : public thrust::unary_function<size_t, ProgramDescriptor>
{
private:
	program_descriptor::Settings* settings;

public:
	thrust::device_ptr<program_descriptor::Settings> settings_ptr;

	create_random(thrust::device_ptr<program_descriptor::Settings> settings_ptr)
		: settings_ptr(settings_ptr)
		, settings(settings_ptr.get())
	{ }

	__host__ __device__
		ProgramDescriptor operator()(size_t index)
	{
		uint32_t seed = hitop::algo::util::hash64_32(index);

		thrust::default_random_engine rng(seed);
		thrust::uniform_int_distribution<size_t> program_length(settings->min_program_size, settings->max_program_size);

		ProgramDescriptor descriptor;
		descriptor.pos = settings->block_size * index;
		descriptor.length = program_length(rng);

		return descriptor;
	}
};

} // namespace program_descriptor

namespace program {

struct fill : public thrust::unary_function<program_descriptor::ProgramDescriptor, uint8_t>
{
	thrust::device_ptr<uint8_t> pool;

	fill(thrust::device_ptr<uint8_t> pool)
		: pool(pool)
	{
		
	}

	__host__ __device__
		uint8_t operator()(program_descriptor::ProgramDescriptor descriptor)
	{
		uint32_t seed = hitop::algo::util::hash64_32(descriptor.pos);

		thrust::default_random_engine rng(seed);
		thrust::uniform_int_distribution<uint8_t> byte_range(0, UINT8_MAX);

		for (size_t i = descriptor.pos;
			 i < descriptor.pos + descriptor.length;
			 i++) {

			pool[i] = byte_range(rng);
		}

		return 0;
	}
};

} // namespace program

}
}

int main(int argc, char* argv[])
{
	hitop::util::AppSettings settings;
	if (!hitop::util::try_parse_settings(argc, argv, settings)) {
		hitop::util::output_usage(std::cerr);
		return -1;
	}

	thrust::host_vector<uint8_t> target_h;
	std::streamsize target_length;
	bool result = try_read_file(settings.input_path, &target_h, &target_length);
	if (!result) {
		return -1;
	}

	if (target_length <= 0) {
		std::cerr << "error: file is empty and cannot be compressed" << std::endl;
		return -1;
	}

	// Actually do stuff
	hitop::util::output_header(std::cout);

	std::cout
		<< "info: started at " << hitop::util::get_timestamp_readable() 
		<< std::endl;

	std::cout
		<< "input file: '" << settings.input_path << std::endl
		<< "size: " << target_length << std::endl;

	// Copy to device
	std::cout
		<< "debug: copying target file to device..."
		<< std::endl;

	thrust::device_vector<char> target_d = target_h;

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// Run the algo
	//

	// Settings
	using namespace hitop::algo;
	using hitop::algo::program_descriptor::ProgramDescriptor;

	const size_t population_count = 100;
	assert(population_count > 0);

	program_descriptor::Settings program_settings_h;
	program_settings_h.min_program_size = target_length * (0.75);
	program_settings_h.max_program_size = target_length;
	program_settings_h.block_size = target_length;

	auto program_settings_d = thrust::device_new<program_descriptor::Settings>();
	hipMemcpy(program_settings_d.get(), &program_settings_h, sizeof(program_descriptor::Settings), hipMemcpyHostToDevice);

	const size_t program_pool_size = population_count * target_length;
	assert(program_pool_size > 0);
	
	// Setup vectors
	const size_t program_descriptors_data_size = sizeof(program_descriptor::ProgramDescriptor) * population_count;

	std::cout
		<< "debug: allocating "
		<< population_count << " program descriptors "
		<< "(" << program_descriptors_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<program_descriptor::ProgramDescriptor> program_descriptors(population_count);

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//

	const size_t program_pool_data_size = sizeof(uint8_t) * program_pool_size;

	std::cout
		<< "debug: allocating program pool "
		<< "(" << program_pool_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<uint8_t> program_pool(program_pool_size);
	
	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	// Initialize program descriptors to random lengths
	std::cout
		<< "debug: initializing program descriptors to random lengths"
		<< std::endl;

	thrust::transform(thrust::counting_iterator<size_t>(0),
					  thrust::counting_iterator<size_t>(program_descriptors.size()),
					  program_descriptors.begin(),
					  program_descriptor::create_random(program_settings_d));
	
	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	// Fill program pool with random values per each program descriptor
	std::cout
		<< "debug: initializing all programs to random data"
		<< std::endl;

	thrust::for_each(program_descriptors.begin(),
					 program_descriptors.end(),
					 program::fill(program_pool.data()));

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// End algo
	//

	// Clear any `device_ptr`s
	thrust::device_delete(program_settings_d);

	std::cout
		<< "info: algorithm ran to completion!" << std::endl
		<< "press any key to exit..." << std::endl;

	std::cin.ignore();
	return 0;
}

