#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include "thrust_ext.cuh"

#include "hitop/util.h"
#include "hitop/algo.cuh"

int main(int argc, char* argv[])
{
	hitop::util::AppSettings settings;
	if (!hitop::util::try_parse_settings(argc, argv, settings)) {
		hitop::util::output_usage(std::cerr);
		return -1;
	}

	thrust::host_vector<uint8_t> target_h;
	std::streamsize target_length;
	bool result = hitop::util::try_read_file(settings.input_path, &target_h, &target_length);
	if (!result) {
		return -1;
	}

	if (target_length <= 0) {
		std::cerr << "error: todo: file is empty and will not be compressed" << std::endl;
		return -1;
	}

	// Actually do stuff
	hitop::util::output_header(std::cout);

	std::cout
		<< "info: started at " << hitop::util::get_timestamp_readable() 
		<< std::endl;

	std::cout
		<< "input file: '" << settings.input_path << std::endl
		<< "size: " << target_length
		<< std::endl
		<< std::endl;

	// Wait for CUDA to initialize
	std::cout
		<< "info: waiting for CUDA to initialize and warm up..."
		<< std::endl;

	hipFree(0);

	std::cout
		<< "info: done"
		<< std::endl
		<< std::endl;

	// Copy to device
	std::cout
		<< "debug: copying target file to device..."
		<< std::endl;

	thrust::device_vector<char> target_d = target_h;

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// Set algorithm Settings
	//

	using namespace hitop;
	using namespace hitop::algo;

	const size_t population_count = 100;
	assert(population_count > 0);

	const size_t selection_elites = 2;

	const bool enable_stats_output = true;
	const size_t generations_per_stats_output = 1;

	const std::streamsize program_pool_size = population_count * target_length;
	assert(program_pool_size > 0);

	size_t generation_num = 0;

	//

	static_assert(sizeof(std::streamsize) == sizeof(algo::buffer_size_t),
				  "Size types are not the same sizes");

	algo::Settings algo_settings_h;
	algo_settings_h.min_program_size = static_cast<algo::buffer_size_t>(target_length * (0.75));
	algo_settings_h.max_program_size = target_length;
	algo_settings_h.block_size = target_length;

	auto algo_settings_d = thrust::device_new<algo::Settings>(algo_settings_h);

	//
	// Allocate device_vectors
	//

	const size_t program_descriptors_data_size = sizeof(program_descriptor::ProgramDescriptor) * population_count;

	std::cout
		<< "debug: allocating "
		<< population_count << " program descriptors "
		<< "(" << program_descriptors_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<program_descriptor::ProgramDescriptor> program_descriptors(population_count);

	thrust::device_vector<program_descriptor::ProgramDescriptor> program_descriptors_temp(population_count);

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//

	const size_t program_pool_data_size = sizeof(uint8_t) * program_pool_size;

	std::cout
		<< "debug: allocating program pools "
		<< "(" << program_pool_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<uint8_t> program_pool(program_pool_size);

	thrust::device_vector<uint8_t> program_pool_temp(program_pool_size);
	
	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//

	const size_t selection_results_data_size = sizeof(SelectionResult) * program_pool_size;

	std::cout
		<< "debug: allocating selection results "
		<< "(" << selection_results_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<SelectionResult> selection_results(program_pool_size);

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// Setup selection settings now that we have allocated everything
	//

	typedef typename algo::selection_method::tournament_selection::Context SelectionContext;

	SelectionContext selection_context_h;
	selection_context_h.program_descriptors = program_descriptors.data().get();
	selection_context_h.program_descriptors_count = program_descriptors.size();
	selection_context_h.rounds = 5;

	auto selection_context_d = thrust::device_new<SelectionContext>(selection_context_h);

	//
	// Initialize initial generation
	//

	// Initialize program descriptors to random lengths
	std::cout
		<< "debug: initializing program descriptors to random lengths"
		<< std::endl;

	thrust::transform(thrust::counting_iterator<size_t>(0),
					  thrust::counting_iterator<size_t>(program_descriptors.size()),
					  program_descriptors.begin(),
					  program_descriptor::create_random(algo_settings_d));
	
	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	// Fill program pool with random values per each program descriptor
	std::cout
		<< "debug: initializing all programs to random data"
		<< std::endl;

	thrust::for_each(program_descriptors.begin(),
					 program_descriptors.end(),
					 program::fill(program_pool.data()));

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	bool is_stop_requested = false;
	while (!is_stop_requested) {

		//
		// Evaluate fitness of initial generation 
		//

		thrust::for_each(program_descriptors.begin(),
						 program_descriptors.end(),
						 program::score(program_pool.data()));

		//
		// Sort program descriptors by their scores descending
		//

		using namespace thrust::placeholders;

		thrust::sort(program_descriptors.begin(),
					 program_descriptors.end(),
					 program_descriptor::greater_score());

		//
		// Calculate statistics on current scores if necessary
		//

		if (enable_stats_output
			&& generation_num % generations_per_stats_output == 0) {

			ProgramDescriptor best = *(program_descriptors.begin());
			ProgramDescriptor worst = *(program_descriptors.end() - 1);

			score_t sum = thrust::transform_reduce(program_descriptors.begin(),
												   program_descriptors.end(),
												   program_descriptor::score(),
												   0.0f,
												   thrust::plus<program_descriptor::score_t>());

			score_t avg = sum / program_descriptors.size();

			std::cout
				<< "gen " << generation_num << ": "
				<< "best = " << best.score << " | "
				<< "avg = " << avg << " | "
				<< "worst = " << worst.score
				<< std::endl;
		}

		//
		// Selection process
		//


		//
		// Select elites if setting specified
		//

		std::cout
			<< "debug: selecting elites..."
			<< std::endl;

		// Keep track of the position to start at in the case that we use elites to skip running the
		// selection method over some of the programs
		auto selection_results_start = selection_results.begin();
		auto program_descriptors_start = program_descriptors.begin();
		auto start_index = 0;

		if (selection_elites > 0) {
			thrust::transform(thrust::counting_iterator<size_t>(0),
							  thrust::counting_iterator<size_t>(selection_elites),
							  selection_results_start,
							  selection_result::from_index());

			start_index += selection_elites;
			thrust::advance(selection_results_start, selection_elites);
			thrust::advance(program_descriptors_start, selection_elites);
		}

		std::cout
			<< "debug: done"
			<< std::endl
			<< std::endl;

		//
		// Run the selection method on the rest
		//

		std::cout
			<< "debug: selecting the rest of population with selection method..."
			<< std::endl;

		thrust::transform(thrust::counting_iterator<size_t>(start_index),
						  thrust::counting_iterator<size_t>(program_descriptors.size()),
						  selection_results_start,
						  selection_method::tournament_selection(selection_context_d));

		std::cout
			<< "debug: done"
			<< std::endl
			<< std::endl;

		//
		// Crossover selection results
		//

		std::cout
			<< "debug: running crossover on selected population with crossover method..."
			<< std::endl;

		thrust::for_each(selection_results.begin(),
						 selection_results.end(),
						 crossover_method::point_crossover(
								 algo_settings_d,
								 program_descriptors.data(), program_pool.data(),
								 program_descriptors_temp.data(), program_pool_temp.data(),
								 program_pool.size()));

		std::cout
			<< "debug: done"
			<< std::endl
			<< std::endl;

		//
		// Mutate newly generated population
		//

		// TODO

		//
		// Swap newly created population with current population vector
		//

		program_descriptors = program_descriptors_temp;
		program_pool = program_pool_temp;

		//
		// Loop back with new generation
		//

		generation_num++;

		if (generation_num >= 100) {
			is_stop_requested = true;
		}
	}

	//
	// End algo
	//

	// Clear any `device_ptr`s
	
	// FIXME: Trying to delete the `thrust::device_ptr` is breaking the build
	//thrust::device_delete(program_settings_d);
	//thrust::device_delete(selection_context_d);
	//thrust::device_delete(random_context_d);

	std::cout
		<< "info: algorithm ran to completion!" << std::endl
		<< "press any key to exit..." << std::endl;

	std::cin.ignore();
	return 0;
}
