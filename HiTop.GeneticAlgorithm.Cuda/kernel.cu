#include <iostream>
#include <iomanip>
#include <fstream>
#include <sstream>
#include <ctime>

#include <thrust/device_new.h>
#include <thrust/device_delete.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/sort.h>
#include <thrust/logical.h>
#include <thrust/random.h>

#include "util.h"

namespace hitop {
namespace algo {

struct Settings
{
	size_t min_program_size;
	size_t max_program_size;
	size_t block_size;
};

namespace util {

__host__ __device__
uint32_t hash32(uint32_t a)
{
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

__host__ __device__
uint64_t hash64(uint64_t key)
{
	key = (~key) + (key << 21); // key = (key << 21) - key - 1;
	key = key ^ (key >> 24);
	key = (key + (key << 3)) + (key << 8); // key * 265
	key = key ^ (key >> 14);
	key = (key + (key << 2)) + (key << 4); // key * 21
	key = key ^ (key >> 28);
	key = key + (key << 31);
	return key;
}

__host__ __device__
uint32_t hash64_32(uint64_t key)
{
	key = (~key) + (key << 18); // key = (key << 18) - key - 1;
	key = key ^ (key >> 31);
	key = key * 21; // key = (key + (key << 2)) + (key << 4);
	key = key ^ (key >> 11);
	key = key + (key << 6);
	key = key ^ (key >> 22);
	return static_cast<uint32_t>(key);
}

} // namespace util

namespace program_descriptor {

typedef float score_t;

struct ProgramDescriptor
{
	size_t pos;
	size_t length;
	score_t score;
};


struct create_random : public thrust::unary_function<size_t, ProgramDescriptor>
{
private:
	algo::Settings* settings;
	thrust::device_ptr<algo::Settings> settings_ptr;

public:
	create_random(thrust::device_ptr<algo::Settings> settings_ptr)
		: settings_ptr(settings_ptr)
		, settings(settings_ptr.get())
	{ }

	__host__ __device__
		ProgramDescriptor operator()(size_t index)
	{
		uint32_t seed = hitop::algo::util::hash64_32(index);

		thrust::default_random_engine rng(seed);
		thrust::uniform_int_distribution<size_t> program_length(settings->min_program_size, settings->max_program_size);

		ProgramDescriptor descriptor;
		descriptor.pos = settings->block_size * index;
		descriptor.length = program_length(rng);
		descriptor.score = NAN;

		return descriptor;
	}
};

struct greater_score : public thrust::binary_function<ProgramDescriptor, ProgramDescriptor, bool>
{
	__host__ __device__
		bool operator()(const ProgramDescriptor& a, const ProgramDescriptor& b)
	{
		return a.score > b.score;
	}
};

struct score : public thrust::unary_function<ProgramDescriptor, score_t>
{
	__host__ __device__
		score_t operator()(const ProgramDescriptor& x)
	{
		return x.score;
	}
};

} // namespace program_descriptor

namespace selection_result {

enum class Mode { Single, Pair };

struct SelectionResult
{
	Mode mode;

	union {
		struct { size_t single_index; };

		struct {
			size_t parent_index_a;
			size_t parent_index_b;
		};
	};
};

__host__ __device__
SelectionResult create_single_index(const size_t index)
{
	SelectionResult result;
	result.mode = Mode::Single;
	result.single_index = index;

	return result;
}

__host__ __device__
SelectionResult create_pair_indices(const size_t parent_index_a, const size_t parent_index_b)
{
	SelectionResult result;
	result.mode = Mode::Pair;
	result.parent_index_a = parent_index_a;
	result.parent_index_b = parent_index_b;

	return result;
}

struct from_index : public thrust::unary_function<size_t, SelectionResult>
{
	__host__ __device__
		SelectionResult operator()(size_t index)
	{
		return create_single_index(index);
	}
};

} // namespace selection_result

namespace program {

struct fill : public thrust::unary_function<program_descriptor::ProgramDescriptor&, void>
{
private:
	thrust::device_ptr<uint8_t> pool;

public:
	fill(thrust::device_ptr<uint8_t> pool)
		: pool(pool)
	{ }

	__host__ __device__
		void operator()(program_descriptor::ProgramDescriptor& descriptor)
	{
		uint32_t seed = hitop::algo::util::hash64_32(descriptor.pos);

		thrust::default_random_engine rng(seed);
		thrust::uniform_int_distribution<uint8_t> byte_range(0, UINT8_MAX);

		for (size_t i = descriptor.pos;
			 i < descriptor.pos + descriptor.length;
			 i++) {

			pool[i] = byte_range(rng);
		}
	}
};

struct score : public thrust::unary_function<program_descriptor::ProgramDescriptor&, void>
{
private:
	thrust::device_ptr<uint8_t> pool;

public:
	score(thrust::device_ptr<uint8_t> pool)
		: pool(pool)
	{
	}

	__host__ __device__
		void operator()(program_descriptor::ProgramDescriptor& descriptor)
	{
		// TODO: Dummy scoring function

		uint32_t seed = hitop::algo::util::hash64_32(descriptor.pos);

		thrust::default_random_engine rng(seed);
		thrust::uniform_real_distribution<float> score_range(0, descriptor.length);

		descriptor.score = score_range(rng);
	}
};


} // namespace program

}
}

int main(int argc, char* argv[])
{
	hitop::util::AppSettings settings;
	if (!hitop::util::try_parse_settings(argc, argv, settings)) {
		hitop::util::output_usage(std::cerr);
		return -1;
	}

	thrust::host_vector<uint8_t> target_h;
	std::streamsize target_length;
	bool result = hitop::util::try_read_file(settings.input_path, &target_h, &target_length);
	if (!result) {
		return -1;
	}

	if (target_length <= 0) {
		std::cerr << "error: todo: file is empty and will not be compressed" << std::endl;
		return -1;
	}

	// Actually do stuff
	hitop::util::output_header(std::cout);

	std::cout
		<< "info: started at " << hitop::util::get_timestamp_readable() 
		<< std::endl;

	std::cout
		<< "input file: '" << settings.input_path << std::endl
		<< "size: " << target_length << std::endl;

	// Wait for CUDA to initialize
	std::cout
		<< "info: waiting for CUDA to initialize and warm up..."
		<< std::endl;

	hipFree(0);

	std::cout
		<< "info: done"
		<< std::endl
		<< std::endl;

	// Copy to device
	std::cout
		<< "debug: copying target file to device..."
		<< std::endl;

	thrust::device_vector<char> target_d = target_h;

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// Set algorithm Settings
	//

	using namespace hitop;
	using namespace hitop::algo;
	using algo::selection_result::SelectionResult;
	using algo::program_descriptor::ProgramDescriptor;

	const size_t population_count = 100;
	assert(population_count > 0);

	const size_t selection_elites = 2;

	const bool enable_stats_output = true;
	const size_t generations_per_stats_output = 1;

	algo::Settings program_settings_h;
	program_settings_h.min_program_size = target_length * (0.75);
	program_settings_h.max_program_size = target_length;
	program_settings_h.block_size = target_length;

	thrust::device_ptr<algo::Settings> program_settings_d = thrust::device_new<algo::Settings>();
	hipMemcpy(program_settings_d.get(), &program_settings_h, sizeof(algo::Settings), hipMemcpyHostToDevice);

	const size_t program_pool_size = population_count * target_length;
	assert(program_pool_size > 0);
	
	size_t generation_num = 0;
	
	//
	// Allocate device_vectors
	//

	const size_t program_descriptors_data_size = sizeof(program_descriptor::ProgramDescriptor) * population_count;

	std::cout
		<< "debug: allocating "
		<< population_count << " program descriptors "
		<< "(" << program_descriptors_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<program_descriptor::ProgramDescriptor> program_descriptors(population_count);

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//

	const size_t program_pool_data_size = sizeof(uint8_t) * program_pool_size;

	std::cout
		<< "debug: allocating program pools "
		<< "(" << program_pool_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<uint8_t> program_pool(program_pool_size);

	thrust::device_vector<uint8_t> program_pool_temp(program_pool_size);
	
	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//

	const size_t selection_results_data_size = sizeof(SelectionResult) * program_pool_size;

	std::cout
		<< "debug: allocating selection results "
		<< "(" << selection_results_data_size << " bytes)..."
		<< std::endl;

	thrust::device_vector<SelectionResult> selection_results(program_pool_size);

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	//
	// Initialize initial generation
	//

	// Initialize program descriptors to random lengths
	std::cout
		<< "debug: initializing program descriptors to random lengths"
		<< std::endl;

	thrust::transform(thrust::counting_iterator<size_t>(0),
					  thrust::counting_iterator<size_t>(program_descriptors.size()),
					  program_descriptors.begin(),
					  program_descriptor::create_random(program_settings_d));
	
	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;

	// Fill program pool with random values per each program descriptor
	std::cout
		<< "debug: initializing all programs to random data"
		<< std::endl;

	thrust::for_each(program_descriptors.begin(),
					 program_descriptors.end(),
					 program::fill(program_pool.data()));

	std::cout
		<< "debug: done"
		<< std::endl
		<< std::endl;


	//
	// Evaluate fitness of initial generation 
	//

	thrust::for_each(program_descriptors.begin(),
					 program_descriptors.end(),
					 program::score(program_pool.data()));

	//
	// Sort program descriptors by their scores descending
	//

	using namespace thrust::placeholders;

	thrust::sort(program_descriptors.begin(),
				 program_descriptors.end(),
				 program_descriptor::greater_score());

	//
	// Calculate statistics on current scores if necessary
	//

	if (enable_stats_output
		&& generation_num % generations_per_stats_output == 0) {

		ProgramDescriptor best = program_descriptors.front();
		ProgramDescriptor worst = program_descriptors.back();

		auto sum = thrust::transform_reduce(program_descriptors.begin(),
					                        program_descriptors.end(),
											program_descriptor::score(),
								            0,
								            thrust::plus<program_descriptor::score_t>());

		auto avg = sum / program_descriptors.size();

		std::cout
			<< "gen " << generation_num << ": "
			<< "best = " << best.score << " | "
			<< "avg = " << avg << " | "
			<< "worst = " << worst.score
			<< std::endl;
	}

	//
	// Selection process
	//


	//
	// Select elites if setting specified
	//

	auto selection_results_start = selection_results.begin();

	if (selection_elites > 0) {
		thrust::transform(thrust::counting_iterator<size_t>(0),
						  thrust::counting_iterator<size_t>(selection_elites),
						  selection_results_start,
						  selection_result::from_index());

		thrust::advance(selection_results_start, selection_elites);
	}

	//
	// Run the selection method on the rest
	//

	// TODO: Finish the rest of this

	//
	// End algo
	//

	// Clear any `device_ptr`s
	thrust::device_delete(program_settings_d);

	std::cout
		<< "info: algorithm ran to completion!" << std::endl
		<< "press any key to exit..." << std::endl;

	std::cin.ignore();
	return 0;
}

