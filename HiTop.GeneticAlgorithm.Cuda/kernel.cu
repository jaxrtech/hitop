#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/transform.h>
#include "thrust_ext.cuh"

#include "spdlog/spdlog.h"

#include "hitop/util.h"
#include "hitop/algo.cuh"

namespace spd = spdlog;

auto logger = spd::stdout_logger_mt("kernel");

int main(int argc, char* argv[])
{
	hitop::util::AppSettings settings;
	if (!hitop::util::try_parse_settings(argc, argv, settings)) {
		hitop::util::output_usage(std::cerr);
		return -1;
	}

	thrust::host_vector<uint8_t> target_h;
	std::streamsize target_length;
	bool result = hitop::util::try_read_file(settings.input_path, &target_h, &target_length);
	if (!result) {
		return -1;
	}

	if (target_length <= 0) {
		logger->error("todo: file is empty and will not be compressed");
		return -1;
	}

	// Actually do stuff
	hitop::util::output_header(std::cout);

	logger->info("started algo");

	logger->debug() << "input file: '" << settings.input_path;
	logger->debug() << "size: " << target_length;

	// Wait for CUDA to initialize
	logger->info("waiting for CUDA to initialize and warm up...");

	hipFree(0);

	logger->info("done");

	// Copy to device
	logger->debug("copying target file to device...");

	thrust::device_vector<char> target_d = target_h;

	logger->debug("done");

	//
	// Set algorithm Settings
	//

	using namespace hitop;
	using namespace hitop::algo;

	const size_t population_count = 100;
	assert(population_count > 0);

	const size_t selection_elites = 2;

	const bool enable_stats_output = true;
	const size_t generations_per_stats_output = 1;

	const std::streamsize program_pool_size = population_count * target_length;
	assert(program_pool_size > 0);

	size_t generation_num = 0;

	//

	static_assert(sizeof(std::streamsize) == sizeof(algo::buffer_size_t),
				  "Size types are not the same sizes");

	algo::Settings algo_settings_h;
	algo_settings_h.min_program_size = static_cast<algo::buffer_size_t>(target_length * (0.75));
	algo_settings_h.max_program_size = target_length;
	algo_settings_h.block_size = target_length;

	auto algo_settings_d = thrust::device_new<algo::Settings>(algo_settings_h);

	//
	// Allocate device_vectors
	//

	const size_t program_descriptors_data_size = sizeof(program_descriptor::ProgramDescriptor) * population_count;

	logger->debug()
		<< "allocating 2 pools of " << population_count << " program descriptors "
		<< "(" << program_descriptors_data_size << " bytes)...";

	thrust::device_vector<program_descriptor::ProgramDescriptor> program_descriptors(population_count);

	thrust::device_vector<program_descriptor::ProgramDescriptor> program_descriptors_temp(population_count);

	logger->debug("done");

	//

	const size_t program_pool_data_size = sizeof(uint8_t) * program_pool_size;

	logger->debug()
		<< "allocating 2 program data pools "
		<< "(" << program_pool_data_size << " bytes each)...";

	thrust::device_vector<uint8_t> program_pool(program_pool_size);

	thrust::device_vector<uint8_t> program_pool_temp(program_pool_size);
	
	logger->debug("done");

	//

	const size_t selection_results_data_size = sizeof(SelectionResult) * program_pool_size;

	logger->debug()
		<< "allocating selection results "
		<< "(" << selection_results_data_size << " bytes)...";

	thrust::device_vector<SelectionResult> selection_results(program_pool_size);

	logger->debug("done");

	//
	// Setup selection settings now that we have allocated everything
	//

	typedef typename algo::selection_method::tournament_selection::Context SelectionContext;

	SelectionContext selection_context_h;
	selection_context_h.program_descriptors = program_descriptors.data().get();
	selection_context_h.program_descriptors_count = program_descriptors.size();
	selection_context_h.rounds = 5;

	auto selection_context_d = thrust::device_new<SelectionContext>(selection_context_h);

	//
	// Initialize initial generation
	//

	// Initialize program descriptors to random lengths
	logger->debug("initializing program descriptors to random lengths");

	thrust::transform(thrust::counting_iterator<size_t>(0),
					  thrust::counting_iterator<size_t>(program_descriptors.size()),
					  program_descriptors.begin(),
					  program_descriptor::create_random(algo_settings_d));
	
	logger->debug("done");

	// Fill program pool with random values per each program descriptor
	logger->debug("initializing all programs to random data");

	thrust::for_each(program_descriptors.begin(),
					 program_descriptors.end(),
					 program::fill(program_pool.data()));

	logger->debug("done");

	bool is_stop_requested = false;
	while (!is_stop_requested) {

		//
		// Evaluate fitness of initial generation 
		//

		thrust::for_each(program_descriptors.begin(),
						 program_descriptors.end(),
						 program::score(program_pool.data()));

		//
		// Sort program descriptors by their scores descending
		//

		using namespace thrust::placeholders;

		thrust::sort(program_descriptors.begin(),
					 program_descriptors.end(),
					 program_descriptor::greater_score());

		//
		// Calculate statistics on current scores if necessary
		//

		if (enable_stats_output
			&& generation_num % generations_per_stats_output == 0) {

			ProgramDescriptor best = *(program_descriptors.begin());
			ProgramDescriptor worst = *(program_descriptors.end() - 1);

			score_t sum = thrust::transform_reduce(program_descriptors.begin(),
												   program_descriptors.end(),
												   program_descriptor::score(),
												   0.0f,
												   thrust::plus<program_descriptor::score_t>());

			score_t avg = sum / program_descriptors.size();

			if (generation_num % generations_per_stats_output == 0)
			{
				logger->notice()
					<< "gen " << generation_num << ": "
					<< "best = " << best.score << " | "
					<< "avg = " << avg << " | "
					<< "worst = " << worst.score;
			}
		}

		//
		// Selection process
		//


		//
		// Select elites if setting specified
		//

		SPDLOG_DEBUG(console, "selecting elites...");

		// Keep track of the position to start at in the case that we use elites to skip running the
		// selection method over some of the programs
		auto selection_results_start = selection_results.begin();
		auto program_descriptors_start = program_descriptors.begin();
		auto start_index = 0;

		if (selection_elites > 0) {
			thrust::transform(thrust::counting_iterator<size_t>(0),
							  thrust::counting_iterator<size_t>(selection_elites),
							  selection_results_start,
							  selection_result::from_index());

			start_index += selection_elites;
			thrust::advance(selection_results_start, selection_elites);
			thrust::advance(program_descriptors_start, selection_elites);
		}

		SPDLOG_DEBUG(console, "done");

		//
		// Run the selection method on the rest
		//

		SPDLOG_DEBUG(console, "selecting the rest of population with selection method...");

		thrust::transform(thrust::counting_iterator<size_t>(start_index),
						  thrust::counting_iterator<size_t>(program_descriptors.size()),
						  selection_results_start,
						  selection_method::tournament_selection(selection_context_d));

		SPDLOG_DEBUG(console, "done");

		//
		// Crossover selection results
		//

		SPDLOG_DEBUG(console, "running crossover on selected population with crossover method...");

		thrust::for_each(selection_results.begin(),
						 selection_results.end(),
						 crossover_method::point_crossover(
								 algo_settings_d,
								 program_descriptors.data(), program_pool.data(),
								 program_descriptors_temp.data(), program_pool_temp.data(),
								 program_pool.size()));


		SPDLOG_DEBUG(console, "done");

		//
		// Mutate newly generated population
		//

		// TODO

		//
		// Swap newly created population with current population vector
		//

		program_descriptors = program_descriptors_temp;
		program_pool = program_pool_temp;

		//
		// Loop back with new generation
		//

		generation_num++;

		if (generation_num >= 100) {
			is_stop_requested = true;
		}
	}

	//
	// End algo
	//

	// Clear any `device_ptr`s
	
	// FIXME: Trying to delete the `thrust::device_ptr` is breaking the build
	//thrust::device_delete(program_settings_d);
	//thrust::device_delete(selection_context_d);
	//thrust::device_delete(random_context_d);

	logger->info("algorithm ran to completion!");
	logger->info("press any key to exit...");

	std::cin.ignore();
	return 0;
}
